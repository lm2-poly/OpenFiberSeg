#include "hip/hip_runtime.h"
// see https://github.com/nosferalatu/SimpleGPUHashTable for hashtable implementation

#include "stdio.h"
#include "stdint.h"
#include "vector"
#include "gpuInterface.h"
#include "iostream"

// 32 bit Murmur3 hash
__device__ uint32_t hash(uint32_t k)
{
    k ^= k >> 16;
    k *= 0x85ebca6b;
    k ^= k >> 13;
    k *= 0xc2b2ae35;
    k ^= k >> 16;
    return k & (kHashTableCapacity - 1);
}

// Create a hash table. For linear probing, this is just an array of KeyValues
KeyValue *create_hashtable()
{
    // Allocate memory
    KeyValue *hashtable;
    hipMalloc(&hashtable, sizeof(KeyValue) * kHashTableCapacity);

    // Initialize hash table to empty
    static_assert(kEmpty == 0xffffffff, "memset expected kEmpty=0xffffffff");
    hipMemset(hashtable, 0xff, sizeof(KeyValue) * kHashTableCapacity);

    return hashtable;
}

// Insert the key/values in kvs into the hashtable
__global__ void gpu_hashtable_insert(KeyValue *hashtable, const KeyValue *kvs, unsigned int numkvs)
{
    unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadid < numkvs)
    {
        uint32_t key = kvs[threadid].key;
        uint32_t value = kvs[threadid].value;
        uint32_t slot = hash(key);

        while (true)
        {
            uint32_t prev = atomicCAS(&hashtable[slot].key, kEmpty, key);
            if (prev == kEmpty || prev == key)
            {
                hashtable[slot].value = value;
                return;
            }

            slot = (slot + 1) & (kHashTableCapacity - 1);
        }
    }
}

void insert_hashtable(KeyValue *pHashTable, const KeyValue *kvs, uint32_t num_kvs)
{
    // Copy the keyvalues to the GPU
    KeyValue *device_kvs;
    hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
    hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

    // Have CUDA calculate the thread block size
    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

    // Create events for GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Insert all the keys into the hash table
    int gridsize = ((uint32_t)num_kvs + threadblocksize - 1) / threadblocksize;
    gpu_hashtable_insert<<<gridsize, threadblocksize>>>(pHashTable, device_kvs, (uint32_t)num_kvs);

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float seconds = milliseconds / 1000.0f;
    printf("    GPU inserted %d items in %f ms (%f million keys/second)\n",
           num_kvs, milliseconds, num_kvs / (double)seconds / 1000000.0f);

    hipFree(device_kvs);
}

// Looks up the  keys in the hashtable, and return the values
__global__ void gpu_hashtable_lookup(KeyValue* hashtable, KeyValue* kvs)
{
    unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadid < kHashTableCapacity)
    {
        uint32_t key = kvs[threadid].key;
        uint32_t slot = hash(key);

        while (true)
        {
            if (hashtable[slot].key == key)
            {
                kvs[threadid].value = hashtable[slot].value;
                return;
            }
            if (hashtable[slot].key == kEmpty)
            {
                kvs[threadid].value = kEmpty;
                return;
            }
            slot = (slot + 1) & (kHashTableCapacity - 1);
        }
    }
}


// Looks up the  keys in the hashtable, and return the values
__global__ void gpu_hashtable_lookup_onArray(KeyValue* hashtable, uint32_t* d_array )
{
    unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadid < kHashTableCapacity)
    {
        uint32_t key = d_array[threadid];
        uint32_t slot = hash(key);

        while (true)
        {
            if (hashtable[slot].key == key)
            {
                d_array[threadid] = hashtable[slot].value;
                return;
            }
            if (hashtable[slot].key == kEmpty)
            {
                d_array[threadid] = kEmpty;
                return;
            }
            slot = (slot + 1) & (kHashTableCapacity - 1);
        }
    }
}

void lookup_hashtable_single_query(KeyValue *hashTable, KeyValue *kvs_query)
{
    // Copy the single keyvalue to the GPU
    KeyValue *device_kvs;
    hipMalloc(&device_kvs, sizeof(KeyValue) );
    hipMemcpy(device_kvs, kvs_query, sizeof(KeyValue) , hipMemcpyHostToDevice);

    gpu_hashtable_lookup<<<1, 1>>>(hashTable, device_kvs);

    hipMemcpy(kvs_query, device_kvs, sizeof(KeyValue) , hipMemcpyDeviceToHost);

    hipFree(device_kvs);
}

void lookup_hashtable_multiple_query(KeyValue* hashTable, KeyValue* kvs_query, uint32_t num_kvs)
{
    // Copy keyvalues to the GPU
    KeyValue *device_kvs;
    hipMalloc(&device_kvs, sizeof(KeyValue)*num_kvs );
    hipMemcpy(device_kvs, kvs_query, sizeof(KeyValue)*num_kvs , hipMemcpyHostToDevice);

    // Have CUDA calculate the thread block size
    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_lookup, 0, 0);

    int myGridSize=num_kvs/threadblocksize+1;

    printf("\ncall to: gpu_hashtable_lookup \n myGridSize: %d\nthreadblocksize: %d\n\n\n",myGridSize, threadblocksize);

    gpu_hashtable_lookup<<<myGridSize, threadblocksize>>>(hashTable, device_kvs);

    hipMemcpy(kvs_query, device_kvs, sizeof(KeyValue)*num_kvs , hipMemcpyDeviceToHost);

    hipFree(device_kvs);
}

void lookup_hashtable_on_array(KeyValue* hashTable, uint32_t* h_array,uint32_t ARRAY_SIZE, uint32_t ARRAY_BYTES)
{
    //copy marker array from host to device
    uint32_t* d_array;
    hipMalloc(&d_array, ARRAY_BYTES );
    hipMemcpy(d_array, h_array, ARRAY_BYTES , hipMemcpyHostToDevice);

    // Have CUDA calculate the thread block size
    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_lookup_onArray, 0, 0);

    int myGridSize=ARRAY_SIZE/threadblocksize+1;

    printf(
        "\ncall to: gpu_hashtable_lookup_onArray \nmyGridSize: %d\nthreadblocksize: %d \nARRAY_SIZE: %d\n\n\n",
        myGridSize, 
        threadblocksize,
        ARRAY_SIZE
        );

    // replace marker keys with corresponding values in the hashtable 
    gpu_hashtable_lookup_onArray<<<myGridSize, threadblocksize>>>(hashTable, d_array);

    hipMemcpy(h_array, d_array, ARRAY_BYTES , hipMemcpyDeviceToHost);

    hipFree(d_array);
}


// Delete each key in kvs from the hash table, if the key exists
// A deleted key is left in the hash table, but its value is set to kEmpty
// Deleted keys are not reused; once a key is assigned a slot, it never moves
__global__ void gpu_hashtable_delete(KeyValue *hashtable, const KeyValue *kvs, unsigned int numkvs)
{
    unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadid < kHashTableCapacity)
    {
        uint32_t key = kvs[threadid].key;
        uint32_t slot = hash(key);

        while (true)
        {
            if (hashtable[slot].key == key)
            {
                hashtable[slot].value = kEmpty;
                return;
            }
            if (hashtable[slot].key == kEmpty)
            {
                return;
            }
            slot = (slot + 1) & (kHashTableCapacity - 1);
        }
    }
}

void delete_hashtable(KeyValue *pHashTable, const KeyValue *kvs, uint32_t num_kvs)
{
    // Copy the keyvalues to the GPU
    KeyValue *device_kvs;
    hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
    hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

    // Have CUDA calculate the thread block size
    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

    // Create events for GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Insert all the keys into the hash table
    int gridsize = ((uint32_t)num_kvs + threadblocksize - 1) / threadblocksize;
    gpu_hashtable_delete<<<gridsize, threadblocksize>>>(pHashTable, device_kvs, (uint32_t)num_kvs);

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float seconds = milliseconds / 1000.0f;
    printf("    GPU delete %d items in %f ms (%f million keys/second)\n",
           num_kvs, milliseconds, num_kvs / (double)seconds / 1000000.0f);

    hipFree(device_kvs);
}

// Iterate over every item in the hashtable; return non-empty key/values
__global__ void gpu_iterate_hashtable(KeyValue *pHashTable, KeyValue *kvs, uint32_t *kvs_size)
{
    unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadid < kHashTableCapacity)
    {
        if (pHashTable[threadid].key != kEmpty)
        {
            uint32_t value = pHashTable[threadid].value;
            if (value != kEmpty)
            {
                uint32_t size = atomicAdd(kvs_size, 1);
                kvs[size] = pHashTable[threadid];
            }
        }
    }
}

// Notes: this one creates an array device_kvs of max size (kNumKeyValues).
// the kernel iterates over entire passed hashtable, and returns non-empty keys
std::vector<KeyValue> iterate_hashtable(KeyValue *pHashTable)
{
    uint32_t *device_num_kvs;
    hipMalloc(&device_num_kvs, sizeof(uint32_t));
    hipMemset(device_num_kvs, 0, sizeof(uint32_t));

    KeyValue *device_kvs;
    hipMalloc(&device_kvs, sizeof(KeyValue) * kNumKeyValues);

    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_iterate_hashtable, 0, 0);

    int gridsize = (kHashTableCapacity + threadblocksize - 1) / threadblocksize;
    gpu_iterate_hashtable<<<gridsize, threadblocksize>>>(pHashTable, device_kvs, device_num_kvs);

    uint32_t num_kvs;
    hipMemcpy(&num_kvs, device_num_kvs, sizeof(uint32_t), hipMemcpyDeviceToHost);

    std::vector<KeyValue> kvs;
    kvs.resize(num_kvs);

    hipMemcpy(kvs.data(), device_kvs, sizeof(KeyValue) * num_kvs, hipMemcpyDeviceToHost);

    hipFree(device_kvs);
    hipFree(device_num_kvs);

    return kvs;
}

// Free the memory of the hashtable
void destroy_hashtable(KeyValue *pHashTable)
{
    hipFree(pHashTable);
}
